﻿#include "Tensor.h"
#include "Common.cuh"
#include <iostream>

using namespace std;
extern hipError_t cudaStatus;

Tensor::Tensor(vector<int> shape, float* data) : shape(shape) {
	// Calculate size & dims
	size = 1;
	dims = 0;
	for (int s : shape) {
		size *= s;
		dims += s;
	}
	// Copy data
	this->data.reserve(size);
	for (int i = 0; i < size; i++) {
		this->data.push_back(data[i]);
	}
	// Upload data onto GPU
	CUDATensor d_data = createCudaTensor();
	HE(hipMemcpy(d_data.data, &(data[0]), size * sizeof(float), hipMemcpyHostToDevice));
	HE(hipMalloc((void**)&(cuda_data), sizeof(CUDATensor)));
	HE(hipMalloc((void**)&(cuda_grad), sizeof(CUDATensor)));
	HE(hipMalloc((void**)&(cuda_sens), sizeof(CUDATensor)));
	HE(hipMemcpy(cuda_data, &d_data, sizeof(CUDATensor), hipMemcpyHostToDevice));
	HE(hipMemcpy(cuda_grad, &createCudaTensor(), sizeof(CUDATensor), hipMemcpyHostToDevice));
	HE(hipMemcpy(cuda_sens, &createCudaTensor(), sizeof(CUDATensor), hipMemcpyHostToDevice));
}

CUDATensor Tensor::createCudaTensor() {
	CUDATensor result;
	HE(hipMalloc((void**)&(result.data), size * sizeof(float)));
	HE(hipMemset(result.data, 0, size * sizeof(float)));
	HE(hipMalloc((void**)&(result.shape), shape.size() * sizeof(int)));
	HE(hipMemcpy(result.shape, &(shape[0]), shape.size() * sizeof(int), hipMemcpyHostToDevice));
	result.dims = dims;
	result.size = size;
	return result;
}

void Tensor::loadData(CUDATensor* src) {
	CUDATensor temp;
	HE(hipMemcpy(&temp, src, sizeof(CUDATensor), hipMemcpyDeviceToHost));
	HE(hipMemcpy(&data[0], temp.data, size * sizeof(float), hipMemcpyDeviceToHost));
}

vector<float> Tensor::getData() {
	loadData(cuda_data);
	return vector<float>(data);
}
vector<float> Tensor::getGrad() {
	loadData(cuda_grad);
	return vector<float>(data);
}

vector<float> Tensor::getSens() {
	loadData(cuda_sens);
	return vector<float>(data);
}	

void Tensor::sync() {
	HE(hipDeviceSynchronize());
}

void Tensor::reset() {
	HE(hipDeviceReset());
}
