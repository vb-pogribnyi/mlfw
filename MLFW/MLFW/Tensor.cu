﻿#include "Tensor.h"
#include "Common.cuh"
#include <iostream>

using namespace std;
extern hipError_t cudaStatus;

Tensor::Tensor(vector<int> shape, float* data) : shape(shape) {
	// Calculate size & dims
	size = 1;
	dims = 0;
	for (int s : shape) {
		size *= s;
		dims += 1;
	}
	// Copy data
	this->data.reserve(size);
	for (int i = 0; i < size; i++) {
		float value = data == 0 ? 0 : data[i];
		this->data.push_back(value);
	}
	// Upload data onto GPU
	CUDATensor d_data = createCudaTensor();
	HE(hipMemcpy(d_data.data, &(this->data[0]), size * sizeof(float), hipMemcpyHostToDevice));
	HE(hipMalloc((void**)&(cuda_data), sizeof(CUDATensor)));
	HE(hipMalloc((void**)&(cuda_grad), sizeof(CUDATensor)));
	HE(hipMalloc((void**)&(cuda_sens), sizeof(CUDATensor)));
	HE(hipMemcpy(cuda_data, &d_data, sizeof(CUDATensor), hipMemcpyHostToDevice));
	HE(hipMemcpy(cuda_grad, &createCudaTensor(), sizeof(CUDATensor), hipMemcpyHostToDevice));
	HE(hipMemcpy(cuda_sens, &createCudaTensor(), sizeof(CUDATensor), hipMemcpyHostToDevice));
}

CUDATensor Tensor::createCudaTensor() {
	CUDATensor result;
	HE(hipMalloc((void**)&(result.data), size * sizeof(float)));
	// TODO: set 1 for the initial grad data
	// TODO: get rid of sensitivity tensor
	HE(hipMemset(result.data, 0, size * sizeof(float)));
	HE(hipMalloc((void**)&(result.shape), shape.size() * sizeof(int)));
	HE(hipMemcpy(result.shape, &(shape[0]), shape.size() * sizeof(int), hipMemcpyHostToDevice));
	result.dims = dims;
	result.size = size;
	return result;
}

void Tensor::downloadData(CUDATensor* src) {
	CUDATensor temp;
	HE(hipMemcpy(&temp, src, sizeof(CUDATensor), hipMemcpyDeviceToHost));
	HE(hipMemcpy(&data[0], temp.data, size * sizeof(float), hipMemcpyDeviceToHost));
}

void Tensor::uploadData(float* data, CUDATensor* dst) {
	CUDATensor temp;
	HE(hipMemcpy(&temp, dst, sizeof(CUDATensor), hipMemcpyDeviceToHost));
	HE(hipMemcpy(temp.data, data, size * sizeof(float), hipMemcpyHostToDevice));
}

vector<float> Tensor::getData() {
	downloadData(cuda_data);
	return vector<float>(data);
}
vector<float> Tensor::getGrad() {
	downloadData(cuda_grad);
	return vector<float>(data);
}

vector<float> Tensor::getSens() {
	downloadData(cuda_sens);
	return vector<float>(data);
}

void Tensor::clear(bool only_grad)
{
	CUDATensor temp;
	if (!only_grad) {
		HE(hipMemcpy(&temp, cuda_data, sizeof(CUDATensor), hipMemcpyDeviceToHost));
		HE(hipMemset(temp.data, 0, size * sizeof(float)));
	}
	HE(hipMemcpy(&temp, cuda_grad, sizeof(CUDATensor), hipMemcpyDeviceToHost));
	HE(hipMemset(temp.data, 0, size * sizeof(float)));
}

vector<int> Tensor::getShape() {
	return shape;
}

int Tensor::getSize() {
	return size;
}

void Tensor::sync() {
	HE(hipDeviceSynchronize());
}

void Tensor::reset() {
	HE(hipDeviceReset());
}

void Tensor::setData(float* data) {
	uploadData(data, cuda_data);
}

void Tensor::setGrad(float* data) {
	uploadData(data, cuda_grad);
}

CUDATensor* Tensor::getCudaData() {
	return cuda_data;
}

CUDATensor* Tensor::getCudaGrad() {
	return cuda_grad;
}

CUDATensor* Tensor::getCudaSens() {
	return cuda_sens;
}
