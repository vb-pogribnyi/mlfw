#include "hip/hip_runtime.h"
﻿#include "MSELoss.h"
#include "../Common.cuh"
#include <iostream>

#define	MSELOSS_PRINT_DEBUG false

using namespace std;
extern hipError_t cudaStatus;

__global__ void mse_loss(CUDATensor* input, CUDATensor* target, CUDATensor* output) {
	extern __shared__ float s[];
	int shared_idx = threadIdx.y * blockDim.x + threadIdx.x;
	int n = blockDim.x * blockDim.y;
	int input_idx = blockIdx.x * n + shared_idx;
	int out_idx = blockIdx.x;
	s[shared_idx] = input->data[input_idx] - target->data[input_idx];
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
		s[0] *= s[0];
#if MSELOSS_PRINT_DEBUG
		printf("%2.3f\n", s[0]);
#endif
		for (int i = 1; i < n; i++) {
			s[0] += s[i] * s[i];
#if MSELOSS_PRINT_DEBUG
			printf("%2.3f\n", s[0]);
#endif
		}
		s[0] /= n;
	}


	output->data[out_idx] = s[0];

}

__global__ void mse_loss_backward(CUDATensor* input, CUDATensor* target, CUDATensor* output) {

#if MSELOSS_PRINT_DEBUG
	printf("Backprop: %2.3f, %2.3f, %i\n", input->data[0], target->data[0], blockDim.y * blockDim.x);
#endif


	//dim3 grid(input_shape[0], 1, 1);
	//dim3 block(input_shape[1], input_shape[2], 1);
	int idx = blockIdx.x * blockDim.y * blockDim.x +
		threadIdx.y * blockDim.x + threadIdx.x;

	output->data[idx] = 2 * (target->data[idx] - input->data[idx]) / (blockDim.y * blockDim.x);
}

MSELoss::MSELoss() {
	// TODO: Initialize loss weights
}

MSELoss::~MSELoss() {
	//
}

void MSELoss::run(Tensor* output, Tensor* input, Tensor* target) {
	record_flow(output, input, target);
	vector<int> input_shape = input->getShape();
	vector<int> output_shape = output->getShape();
	dim3 grid(input_shape[0], 1, 1);
	dim3 block(input_shape[1], input_shape[2], 1);
	int shared_mem_items = input_shape[1] + input_shape[2];
	mse_loss << <grid, block, sizeof(float) * shared_mem_items >> > (input->getCudaData(),
		target->getCudaData(),
		output->getCudaData());
	HE(hipPeekAtLastError());
}

void MSELoss::update(float lr) {
	//
}

void MSELoss::propagate() {
	// No changes to weights
	// (a2(a1 * X + b1) + b2 - T)^2 -> 0
	// 2 * E * I2
	// 2 * E * a2 * X

	// Set input vector gradient to 2 * E, twice the output
	vector<int> input_shape = flow_input1->getShape();
	dim3 grid(input_shape[0], 1, 1);
	dim3 block(input_shape[1], input_shape[2], 1);
	mse_loss_backward << <grid, block >> > (flow_input1->getCudaData(),
		flow_input2->getCudaData(), 
		flow_input1->getCudaGrad());
	HE(hipPeekAtLastError());
}
