#include "hip/hip_runtime.h"
﻿#include "Conv.h"
#include "../Common.cuh"
#include <iostream>

#define RUN_CONV(function, offset, limits, weight_shape, output_shape, parameters) \
	for (offset.example = 0; offset.example < output_shape[0]; offset.example += limits.example) { \
		for (offset.ch_in = 0; offset.ch_in < weight_shape[0]; offset.ch_in += limits.ch_in) { \
			for (offset.ch_out = 0; offset.ch_out < weight_shape[1]; offset.ch_out += limits.ch_out) { \
				for (offset.x_out = 0; offset.x_out < output_shape[2]; offset.x_out += limits.x_out) { \
					for (offset.x_in = 0; offset.x_in < weight_shape[2]; offset.x_in += limits.x_in) { \
						int grid_x = min(output_shape[2] - offset.x_out, limits.x_out); \
						int grid_y = 1; \
						int grid_z = min(output_shape[0] - offset.example, limits.example); \
						dim3 grid(grid_x, grid_y, grid_z); \
						int block_x = min(weight_shape[2] - offset.x_in, limits.x_in); \
						int block_y = 1; \
						int block_z = min(weight_shape[0] - offset.ch_in, limits.ch_in) * \
							min(weight_shape[1] - offset.ch_out, limits.ch_out); \
						dim3 block(block_x, block_y, block_z); \
						HE(hipMemcpy(d_offset, &offset, sizeof(ConvOffset), hipMemcpyHostToDevice)); \
						function <<<grid, block>>> parameters; \
						HE(hipPeekAtLastError()); \
					} \
				} \
			} \
		} \
	} \

#define	CONV_PRINT_DEBUG false
#define CONV_BACK_PRINT_DEBUG false

using namespace std;
extern hipError_t cudaStatus;

__device__ ConvInfo get_indices(CUDATensor* input, CUDATensor* output, CUDATensor* weight, ConvOffset* offset) {
	ConvInfo result = { 0 };
	int in_width = 1;
	int in_height = 1;
	result.out_width = 1;
	result.out_height = 1;
	result.n_examples = input->shape[0];
	result.n_channels_in = weight->shape[0];
	result.n_channels_out = weight->shape[1];
	if (input->dims == 4) {
		in_width = input->shape[input->dims - 2];
		in_height = input->shape[input->dims - 1];
		result.out_width = output->shape[output->dims - 2];
		result.out_height = output->shape[output->dims - 1];
	}
	else {
		in_width = input->shape[input->dims - 1];
		result.out_width = output->shape[output->dims - 1];
	}
	int x_out = offset->x_out + blockIdx.x;
	int y_out = offset->y_out + blockIdx.y;
	int x_in = offset->x_in + threadIdx.x;
	int y_in = offset->y_in + threadIdx.y;

	int channel = offset->ch_in * result.n_channels_out + offset->ch_out + threadIdx.z;
	int ch_in = channel / result.n_channels_out;
	int ch_out = channel % result.n_channels_out;
	int example = offset->example + blockIdx.z;

	result.in_idx = example * result.n_channels_in * in_width * in_height +						// example
		ch_in * in_height * in_width +															// in channel
		(y_out + y_in) * in_width +																// height
		x_out + x_in;																			// width

	result.kern_idx = ch_in * weight->shape[2] * weight->shape[3] +								// in channel
		ch_out * weight->shape[0] * weight->shape[2] * weight->shape[3] +						// out channel
		y_in * weight->shape[3] +																// height
		x_in;																					// width

	result.out_idx = example * result.n_channels_out * result.out_width * result.out_height +	// example
		ch_out * result.out_width * result.out_height +											// out channel
		y_out * result.out_width + 																// height
		x_out;																					// width
	result.bias_idx = ch_out;

	return result;
}

__global__ void convolve(CUDATensor* input, CUDATensor* output, CUDATensor* weight, CUDATensor* bias, ConvOffset* offset) {
	ConvInfo indices = get_indices(input, output, weight, offset);

	atomicAdd(output->data + indices.out_idx, input->data[indices.in_idx] * weight->data[indices.kern_idx]);

#if CONV_PRINT_DEBUG
	printf("Thread idx: %i, offset_in: %i, offset_out: %i, channel: %i\n",
		threadIdx.z, offset->ch_in, offset->ch_out, channel);
	printf("example: %i, ch_out: %i, ch_in: %i, in_idx: %i, kern_idx: %i, bias_idx: %i, out_idx: %i\n",
		example, ch_out, ch_in, in_idx, kern_idx, bias_idx, out_idx);
	printf("Output: %2.5f\n",
		output->data[out_idx]);
#endif
}

__global__ void add_bias(CUDATensor* input, CUDATensor* output, CUDATensor* weight, CUDATensor* bias, ConvOffset* offset) {
	ConvInfo indices = get_indices(input, output, weight, offset);

	atomicAdd(output->data + indices.out_idx, bias->data[indices.bias_idx]);
#if CONV_PRINT_DEBUG
	printf("BIAS:\n");
	printf("Out idx: %i, bias idx: %i, output: %2.3f\n", out_idx, bias_idx, output->data[out_idx]);
#endif
}

__global__ void convolve_backward(CUDATensor* input, CUDATensor* d_input, CUDATensor* d_output,
	CUDATensor* weight, CUDATensor* bias, 
	CUDATensor* d_weight, CUDATensor* d_bias, 
	ConvOffset* offset) {
	ConvInfo indices = get_indices(input, d_output, weight, offset);
	int n_vals_w = indices.out_width * indices.out_height * indices.n_examples * indices.n_channels_out;
	int n_vals_b = indices.out_width * indices.out_height * indices.n_examples * indices.n_channels_out * 
		indices.n_channels_in * weight->shape[2] * weight->shape[3];

	atomicAdd(d_weight->data + indices.kern_idx, input->data[indices.in_idx] * d_output->data[indices.out_idx] / n_vals_w);
	atomicAdd(d_bias->data + indices.bias_idx, d_output->data[indices.out_idx] / n_vals_b);

	atomicAdd(d_input->data + indices.in_idx, d_output->data[indices.out_idx] * weight->data[indices.kern_idx] / n_vals_w);

#if CONV_BACK_PRINT_DEBUG
		printf("Width: %i, height: %i, in idx: %i, out idx: %i, w_idx: %i, b_idx: %i\n", in_width, in_height, in_idx, out_idx, w_idx, b_idx);
		printf("Input: %2.3f, output grad: %2.3f, values affected by W: %i, values affected by B: %i\n", input->data[in_idx], d_output->data[out_idx], n_vals_w, n_vals_b);
		printf("New dw: %2.5f, new db: %2.5f, w_idx: %i, b_idx: %i\n", d_weight->data[w_idx], d_bias->data[b_idx], w_idx, b_idx);
		printf("Sens: %2.5f, weight: %2.3f, grad: %2.3f, in_idx: %i, out_idx: %i, w_idx: %i\n", d_input->data[in_idx], weight->data[w_idx], d_output->data[out_idx], in_idx, out_idx, w_idx);
#endif
}


Conv1d::Conv1d(const int ch_in, const int ch_out, const int width) : weight(0), bias(0) {
	limits.ch_in = 1;
	limits.ch_out = 1;
	limits.example = 1;
	limits.x_in = 1;
	limits.y_in = 1;
	limits.x_out = 1;
	limits.y_out = 1;

	limits.ch_in = 2;
	limits.ch_out = 2;
	limits.example = 16;
	limits.x_in = 8;
	limits.y_in = 8;
	limits.x_out = 16;
	limits.y_out = 16;

	vector<int> weight_shape = { ch_in, ch_out, width, 1 };
	vector<int> bias_shape = { ch_out };
	vector<float> weight_data(ch_in * ch_out * width, 0);
	vector<float> bias_data(ch_out, 0);

	weight = new Tensor(weight_shape, &weight_data[0]);
	bias = new Tensor(bias_shape, &bias_data[0]);
}

Conv1d::~Conv1d() {
	if (weight) {
		delete weight;
	}
	if (bias) {
		delete bias;
	}
}

void Conv1d::checkShapes(vector<int> input_shape, vector<int> output_shape, vector<int> weight_shape) {
	int exp_out_width = input_shape[2] - 2 * (weight_shape[2] / 2);
	int exp_out_channels = weight_shape[1];
	int exp_in_channels = weight_shape[0];
	if (input_shape[1] != exp_in_channels)
		throw TensorShapeError();
	if (output_shape[1] != exp_out_channels)
		throw TensorShapeError();
	if (exp_out_width <= 0 || output_shape[2] != exp_out_width)
		throw TensorShapeError();

}

void Conv1d::run(Tensor* output, Tensor* input, Tensor* _) {
	record_flow(output, input);
	output->clear();
	vector<int> input_shape = input->getShape();
	vector<int> output_shape = output->getShape();
	vector<int> weight_shape = weight->getShape();
	// throws TensorShapeError
	checkShapes(input_shape, output_shape, weight_shape);
	ConvOffset offset = { 0 };
	ConvOffset* d_offset;
	HE(hipMalloc((void**)&(d_offset), sizeof(ConvOffset)));

	RUN_CONV(convolve, offset, limits, weight_shape, output_shape, (
		input->getCudaData(),
		output->getCudaData(),
		weight->getCudaData(),
		bias->getCudaData(),
		d_offset
		)
	);
	auto limits_local = limits;
	limits_local.x_in = 1;
	limits_local.ch_in = 1;
	weight_shape[0] = 1;
	weight_shape[2] = 1;
	RUN_CONV(add_bias, offset, limits_local, weight_shape, output_shape, (
		input->getCudaData(),
		output->getCudaData(),
		weight->getCudaData(),
		bias->getCudaData(),
		d_offset
		)
	);
}

void Conv1d::update(float lr) {
	//
}

void Conv1d::propagate() {
	// Out = f(Wx + b)
	// dOut/dW = df/d(Wx + b) * x

	weight->clear(true);
	bias->clear(true);
	vector<int> input_shape = flow_input1->getShape();
	vector<int> output_shape = flow_output->getShape();
	vector<int> weight_shape = weight->getShape();

	ConvOffset offset = { 0 };
	ConvOffset* d_offset;

	HE(hipMalloc((void**)&(d_offset), sizeof(ConvOffset)));

	RUN_CONV(convolve_backward, offset, limits, weight_shape, output_shape, (
		flow_input1->getCudaData(),
		flow_input1->getCudaGrad(),
		flow_output->getCudaGrad(),
		weight->getCudaData(),
		bias->getCudaData(),
		weight->getCudaGrad(),
		bias->getCudaGrad(),
		d_offset)
	);
}
